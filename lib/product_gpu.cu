#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

extern "C" {
#include "../include/formats.h"
#include "../include/product_gpu.h"
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// ######################################## csr ############################################

/*
template <unsigned int THD> __global__ void product_one_row_one_block_csr(int M, int* irp, int* ja, double* as, double* array, double* result) {

   //block sum
	__shared__ double block_sum[THD];
   
   int tid = threadIdx.x;
   int row = blockIdx.x;
   int i;

   block_sum[tid] = 0;
   //check row
   if (row < M) {

      //check thread
      int limit = irp[row+1];

      // block sums row
      double sum = 0;
      for (i = irp[row] + tid; i < limit; i += blockDim.x)
         sum += as[i] * array[ja[i]];
      
      block_sum[tid] = sum;
      //wait for all to finish
      __syncthreads();

      //block reduction (max efficency)
      if (THD >= 1024) { if (tid < 512) { block_sum[tid] += block_sum[tid + 512]; } __syncthreads(); }

      if (THD >= 512) { if (tid < 256) { block_sum[tid] += block_sum[tid + 256]; } __syncthreads(); }

      if (THD >= 256) { if (tid < 128) { block_sum[tid] += block_sum[tid + 128]; } __syncthreads(); }

      if (THD >= 128) { if (tid < 64) { block_sum[tid] += block_sum[tid + 64]; } __syncthreads(); }

      if (THD >= 64) { if (tid < 32) { block_sum[tid] += block_sum[tid + 32]; } __syncthreads(); }

      //last warp
      if (tid < 16) { block_sum[tid] += block_sum[tid + 16]; __syncwarp(); }
      if (tid < 8) { block_sum[tid] += block_sum[tid + 8]; __syncwarp(); }
      if (tid < 4) { block_sum[tid] += block_sum[tid + 4]; __syncwarp(); }
      if (tid < 2) { block_sum[tid] += block_sum[tid + 2]; __syncwarp(); }
      if (tid < 1) { block_sum[tid] += block_sum[tid + 1]; __syncwarp(); }

      //reduction over, save result in final array
      if (tid == 0)
         result[row] = block_sum[tid];

   }

}
*/
__global__ void product_one_row_one_warp_csr(unsigned int M, unsigned int* __restrict__ irp, unsigned int* __restrict__ ja, double*  __restrict__ as, double* __restrict__ array, double* __restrict__ result) {

   __shared__ double warp_sum[BLOCK_SIZE];

   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   int b_tid = threadIdx.x;
   int warp_id = tid % WARP_SIZE;
   int row = tid / WARP_SIZE;
   int i;

   warp_sum[b_tid] = 0;
   if (row < M) {

      // warp sums row
      int limit = irp[row+1];
      double sum = 0;
      for (i = irp[row] + warp_id; i < limit; i += WARP_SIZE) {
         sum += as[i] * array[ja[i]];
      }

      warp_sum[b_tid] = sum;
      
      __syncwarp();

      // we have only one warp, no need to sync entire block
      if (warp_id < 16) { warp_sum[b_tid] += warp_sum[b_tid + 16]; __syncwarp();}
      
      if (warp_id < 8) { warp_sum[b_tid] += warp_sum[b_tid + 8]; __syncwarp();}

      if (warp_id < 4) { warp_sum[b_tid] += warp_sum[b_tid + 4]; __syncwarp();}
      
      if (warp_id < 2) { warp_sum[b_tid] += warp_sum[b_tid + 2]; __syncwarp();}
      
      if (warp_id < 1) { warp_sum[b_tid] += warp_sum[b_tid + 1]; __syncwarp();}
      
      //reduction over, save result in final array
      if (warp_id == 0) {
         result[row] = warp_sum[b_tid];
      }

   }


}
/*
template <unsigned int N> __global__ void product_one_row_N_warp_csr(unsigned int M, unsigned int* __restrict__ irp, unsigned int* __restrict__ ja, double*  __restrict__ as, double* __restrict__ array, double* __restrict__ result) {

   __shared__ double warp_sum[BLOCK_SIZE];

   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   int b_tid = threadIdx.x;
   int warp_id = tid % (WARP_SIZE*2);
   int row = tid / (WARP_SIZE*N);
   int i;

   warp_sum[b_tid] = 0;
   if (row < M) {

      // warp sums row
      int limit = irp[row+1];
      double sum = 0;
      for (i = irp[row] + warp_id; i < limit; i += WARP_SIZE) {
         sum += as[i] * array[ja[i]];
      }

      warp_sum[b_tid] = sum;
      
      __syncthreads();

      //block reduction (max efficency)
      if (N >= 32) { if (tid < 512) { warp_sum[b_tid] += warp_sum[b_tid + 512]; } __syncthreads(); }

      if (N >= 16) { if (tid < 256) { warp_sum[b_tid] += warp_sum[b_tid + 256]; } __syncthreads(); }

      if (N >= 8) { if (tid < 128) { warp_sum[b_tid] += warp_sum[b_tid + 128]; } __syncthreads(); }

      if (N >= 4) { if (tid < 64) { warp_sum[b_tid] += warp_sum[b_tid + 64]; } __syncthreads(); }

      if (N >= 2) { if (tid < 32) { warp_sum[b_tid] += warp_sum[b_tid + 32]; } __syncthreads(); }


      // we have only one warp, no need to sync entire block
      if (warp_id < 16) { warp_sum[b_tid] += warp_sum[b_tid + 16]; __syncwarp();}
      
      if (warp_id < 8) { warp_sum[b_tid] += warp_sum[b_tid + 8]; __syncwarp();}

      if (warp_id < 4) { warp_sum[b_tid] += warp_sum[b_tid + 4]; __syncwarp();}
      
      if (warp_id < 2) { warp_sum[b_tid] += warp_sum[b_tid + 2]; __syncwarp();}
      
      if (warp_id < 1) { warp_sum[b_tid] += warp_sum[b_tid + 1]; __syncwarp();}
      
      //reduction over, save result in final array
      if (warp_id == 0) {
         result[row] = warp_sum[b_tid];
      }

   }


} */

/*
__global__ void product_one_row_one_thread_csr(int M, int* irp, int* ja, double* as, double* array, double* result) {

   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   int row = tid;
   int i;
   //printf("%d\n", tid);

   if (row < M) {

      // warp sums row
      int limit = irp[row+1];
      double sum = 0;
      for (i = irp[row]; i < limit; i += 1) {
         sum += as[i] * array[ja[i]];
      }

      result[row] = sum;
      
   }


}

template <unsigned int N> __global__ void product_N_row_one_thread_csr(int M, int* irp, int* ja, double* as, double* array, double* result) {

   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   int row_start = tid*N;
   int row_end = tid*N + N;
   int i;
   //printf("%d\n", tid);

   // warp sums row
   for (; row_start < row_end && row_start < M; row_start++) {
      double sum = 0;
      int limit = irp[row_start+1];
      for (i = irp[row_start]; i < limit; i += 1) {
         sum += as[i] * array[ja[i]];
      }

      result[row_start] = sum;
   }

}
*/

extern "C"
float cuda_product_csr(csr_matrix* matrix, double* array, double* result) {

   unsigned int* irp_gpu;
   unsigned int* ja_gpu;
   double* as_gpu;
   double* array_gpu;
   double* result_gpu;

   //alloc arguments
   gpuErrchk( hipMalloc((void**) &irp_gpu, sizeof(int)*(matrix->M+1)) );
   gpuErrchk( hipMalloc((void**) &ja_gpu, sizeof(int)*matrix->nz) );
   gpuErrchk( hipMalloc((void**) &as_gpu, sizeof(double)*matrix->nz) );
   gpuErrchk( hipMalloc((void**) &array_gpu, sizeof(double)*matrix->M) );
   gpuErrchk( hipMalloc((void**) &result_gpu, sizeof(double)*matrix->M) );

   //copy arguments
   gpuErrchk( hipMemcpy(irp_gpu, matrix->irp, sizeof(int)*(matrix->M+1), hipMemcpyHostToDevice) );
   gpuErrchk( hipMemcpy(ja_gpu, matrix->ja, sizeof(int)*matrix->nz, hipMemcpyHostToDevice) );
   gpuErrchk( hipMemcpy(as_gpu, matrix->as, sizeof(double)*matrix->nz, hipMemcpyHostToDevice) );
   gpuErrchk( hipMemcpy(array_gpu, array, sizeof(double)*matrix->M, hipMemcpyHostToDevice) );
   
   //set up timer
   hipEvent_t start, stop;
   gpuErrchk( hipEventCreate(&start) );
   gpuErrchk( hipEventCreate(&stop) );

   gpuErrchk( hipEventRecord(start, 0) );
   product_one_row_one_warp_csr<<<(matrix->M*WARP_SIZE) / BLOCK_SIZE + 1, BLOCK_SIZE , sizeof(double)*BLOCK_SIZE>>>(matrix->M, irp_gpu, ja_gpu, as_gpu, array_gpu, result_gpu);      
   gpuErrchk( hipEventRecord(stop, 0) );

   gpuErrchk( hipEventSynchronize(stop) );

   //calculate time of computation (in ms)
   float time;
   gpuErrchk( hipEventElapsedTime(&time, start, stop) );
   
   //copy result back from gpu
   gpuErrchk( hipMemcpy(result, result_gpu, sizeof(double)*matrix->M, hipMemcpyDeviceToHost) );
   
   //free everything
   gpuErrchk( hipFree(irp_gpu) );
   gpuErrchk( hipFree(ja_gpu) );
   gpuErrchk( hipFree(as_gpu) );
   gpuErrchk( hipFree(result_gpu) );
   gpuErrchk( hipFree(array_gpu) );
   gpuErrchk( hipEventDestroy(start) );
   gpuErrchk( hipEventDestroy(stop) );

   return time;

}

// ################################ ellpack #########################################

__global__ void product_one_row_one_warp_ellpack(unsigned int M, unsigned int maxnz, unsigned int* __restrict__ ja, double* __restrict__ as, double* __restrict__ array, double* __restrict__ result) {

   __shared__ double warp_sum[BLOCK_SIZE];

   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   int b_tid = threadIdx.x;
   int warp_id = tid % WARP_SIZE;
   unsigned int row = tid / WARP_SIZE;
   int i;

   warp_sum[b_tid] = 0;
   if (row < M) {

      // warp sums row
      double sum = 0;
      int index = row*maxnz;
      for (i = warp_id; i < maxnz; i += WARP_SIZE) {
         sum += as[index + i] * array[ja[index + i]];
      }

      warp_sum[b_tid] = sum;
      
      __syncwarp();

      // we have only one warp, no need to sync entire block
      if (warp_id < 16) { warp_sum[b_tid] += warp_sum[b_tid + 16]; __syncwarp();}
      
      if (warp_id < 8) { warp_sum[b_tid] += warp_sum[b_tid + 8]; __syncwarp();}

      if (warp_id < 4) { warp_sum[b_tid] += warp_sum[b_tid + 4]; __syncwarp();}
      
      if (warp_id < 2) { warp_sum[b_tid] += warp_sum[b_tid + 2]; __syncwarp();}
      
      if (warp_id < 1) { warp_sum[b_tid] += warp_sum[b_tid + 1]; __syncwarp();}
      
      //reduction over, save result in final array
      if (warp_id == 0) {
         result[row] = warp_sum[b_tid];
      }

   }


}

extern "C"
float cuda_product_ellpack(ellpack_matrix* matrix, double* array, double* result) {

   unsigned int* ja_gpu;
   double* as_gpu;
   double* array_gpu;
   double* result_gpu;

   //alloc arguments
   gpuErrchk( hipMalloc((void**) &ja_gpu, (unsigned long)sizeof(unsigned int)*matrix->M*matrix->maxnz) );
   gpuErrchk( hipMalloc((void**) &as_gpu, (unsigned long)sizeof(double)*matrix->M*matrix->maxnz) );
   gpuErrchk( hipMalloc((void**) &array_gpu, sizeof(double)*matrix->M) );
   gpuErrchk( hipMalloc((void**) &result_gpu, sizeof(double)*matrix->M) );

   //copy arguments
   gpuErrchk( hipMemcpy(ja_gpu, matrix->ja, (unsigned long) sizeof(int)*matrix->M*matrix->maxnz, hipMemcpyHostToDevice) );
   gpuErrchk( hipMemcpy(as_gpu, matrix->as, (unsigned long) sizeof(double)*matrix->M*matrix->maxnz, hipMemcpyHostToDevice) );
   gpuErrchk( hipMemcpy(array_gpu, array, sizeof(double)*matrix->M, hipMemcpyHostToDevice) );
   
   //set up timer
   hipEvent_t start, stop;
   gpuErrchk( hipEventCreate(&start) );
   gpuErrchk( hipEventCreate(&stop) );

   gpuErrchk( hipEventRecord(start, 0) );
   product_one_row_one_warp_ellpack<<<(matrix->M*WARP_SIZE) / BLOCK_SIZE + 1, BLOCK_SIZE , sizeof(double)*BLOCK_SIZE>>>(matrix->M, matrix->maxnz, ja_gpu, as_gpu, array_gpu, result_gpu);      
   gpuErrchk( hipEventRecord(stop, 0) );

   gpuErrchk( hipEventSynchronize(stop) );

   //calculate time of computation (in ms)
   float time;
   gpuErrchk( hipEventElapsedTime(&time, start, stop) );
   
   //copy result back from gpu
   gpuErrchk( hipMemcpy(result, result_gpu, sizeof(double)*matrix->M, hipMemcpyDeviceToHost) );
   
   //free everything
   gpuErrchk( hipFree(ja_gpu) );
   gpuErrchk( hipFree(as_gpu) );
   gpuErrchk( hipFree(result_gpu) );
   gpuErrchk( hipFree(array_gpu) );
   gpuErrchk( hipEventDestroy(start) );
   gpuErrchk( hipEventDestroy(stop) );

	return time;

}